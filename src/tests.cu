#include "util.hpp"
#include "genasm_gpu.hpp"
#include "genasm_cpu.hpp"
#include "bitvector_test.hpp"

#include <iostream>
#include <math.h>
#include <fstream>
#include <string>
#include <algorithm>
#include <chrono>
#include <iomanip>

using namespace std;

#ifdef __GNUC__
    #include <experimental/filesystem>
    using namespace std::experimental;
#else
    #include <filesystem>
#endif


bool enable_log = true;
string optimization_blocker = "";

bool cigarFormatCorrect(Alignment_t &alignment){
    stringstream cigar_ss(alignment.cigar);
    cigar_ss.peek(); //set the eof bit
    while(!cigar_ss.eof()){
        unsigned int edit_count;
        char edit_type;

        cigar_ss >> edit_count;
        cigar_ss >> edit_type;
        cigar_ss.peek(); //set the eof bit

        if(cigar_ss.fail()){
            cout << "CIGAR had bad format" << endl;
            return false;
        }

        if(edit_count == 0){
            cout << "CIGAR cannot contain edits with count 0" << endl;
            return false;
        }

        if(edit_type == 'I'){
        }
        else if(edit_type == 'D'){
        }
        else if(edit_type == 'X' || edit_type == '=' || edit_type == 'M'){
        }
        else{
            cout << "CIGAR contains unknown edit type '" << edit_type << "'" << endl;
            return false;
        }
    }
    return true;
}

bool cigarInBoundsAndCoversRead(Alignment_t &alignment, CandidateLocation_t &location, Read_t &read, Genome_t &reference){
    long long i = location.start_in_reference;
    long long j = 0;

    stringstream cigar_ss(alignment.cigar);
    cigar_ss.peek(); //set the eof bit
    while(!cigar_ss.eof()){
        unsigned int edit_count;
        char edit_type;
        cigar_ss >> edit_count;
        cigar_ss >> edit_type;
        
        cigar_ss.peek(); //set the eof bit

        if(edit_type == 'I'){
            j += edit_count;
        }
        else if(edit_type == 'D'){
            i += edit_count;
        }
        else{
            i += edit_count;
            j += edit_count;
        }
    }

    if(j < read.content.size()){
        cout << "CIGAR didn't cover entire read" << endl;
        return false;
    }

    if(j > read.content.size()){
        cout << "CIGAR went out of bounds of read" << endl;
        return false;
    }

    if(i > reference.content.size()){
        cout << "CIGAR went out of bounds of reference" << endl;
        return false;
    }

    return true;
}

bool validateCigarString(Alignment_t &alignment, CandidateLocation_t &location, Read_t &read, Genome_t &reference){
    /*
     * test if the given CIGAR string is a correct transformation from reference to read
     * return true if correct, false otherwise, and print potential error messages
    */
    if(!cigarFormatCorrect(alignment)){
        cout << "CIGAR format wrong" << endl;
        return false;
    }
    if(!cigarInBoundsAndCoversRead(alignment, location, read, reference)){
        cout << "CIGAR runs out of bounds or too short to cover read" << endl;
        return false;
    }

    long long i = location.start_in_reference;
    long long j = 0;
    long long edits_in_cigar_string = 0;

    stringstream cigar_ss(alignment.cigar);
    cigar_ss.peek(); //would set the eof bit for the empty string
    while(!cigar_ss.eof()){
        unsigned int edit_count;
        char edit_type;
        cigar_ss >> edit_count;
        cigar_ss >> edit_type;
        cigar_ss.peek(); //set the eof bit

        if(edit_type == 'I'){
            j += edit_count;
            edits_in_cigar_string += edit_count;
        }
        else if(edit_type == 'D'){
            i += edit_count;
            edits_in_cigar_string += edit_count;
        }
        else{ //M, X, =
            for(int e = 0; e < edit_count; e++){
                if(edit_type == 'X' && cigar_char_equals(reference.content[i], read.content[j])){
                    cout << "CIGAR contains 'X' but reference[i] and read[j] match" << endl;
                    return false;
                }
                if(edit_type == '=' && !cigar_char_equals(reference.content[i], read.content[j])){
                    cout << "CIGAR contains '=' but reference[i] and read[j] mismatch" << endl;
                    return false;
                }
                if(edit_type == 'M' && reference.content[i] != read.content[j]){
                    edits_in_cigar_string++;
                }
                i++;
                j++;
            }
            if(edit_type == 'X'){
                edits_in_cigar_string += edit_count;
            }
        }
    }

    if(edits_in_cigar_string != alignment.edit_distance){
        cout << "CIGAR has " << edits_in_cigar_string << " edits, while the reported edit disatance is " << alignment.edit_distance << endl;
        return false;
    }

    return true;
}

void gpu_algorithm_correctness_test(){
    Genome_t reference;
    reference.content = "AAAACCCCGGGGTTTT";
    
    CandidateLocation_t ref_begin;
    ref_begin.start_in_reference = 0;
    ref_begin.start_in_chromosome = 0;
    ref_begin.strand = true;
    ref_begin.chromosome = "";
    vector<CandidateLocation_t> ref_begin_vec(1, ref_begin);

    vector<Read_t> reads;
    reads.push_back({"test_read_4d12m4i",           "CCCCGGGGTTTTAAAA",         ref_begin_vec});
    reads.push_back({"test_read_16m",               "AAAACCCCGGGGTTTT",         ref_begin_vec});
    reads.push_back({"test_read_3d7m",              "ACCCCGG",                  ref_begin_vec});
    reads.push_back({"test_read_4m4d4m4i4m",        "AAAAGGGGAAAATTTT",         ref_begin_vec});
    reads.push_back({"test_read_12s4m",             "AAAAAAAAAAAAAAAA",         ref_begin_vec});
    reads.push_back({"test_read_1m1s1i3m1s2m3i",    "ATTAACGCCTTT",             ref_begin_vec});
    reads.push_back({"test_read_oversized",         "TTTTAAAACCCCGGGGTTTTAAAA", ref_begin_vec});
    reads.push_back({"test_read_empty",             "",                         ref_begin_vec});
    reads.push_back({"test_read_len64",             "TTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTAAAACCCCGGGGTTTTAAAA", ref_begin_vec});

    //vector<int> correct_edit_distances = {8, 0, 3, 8, 12, 6, 8, 0};
    vector<int> correct_edit_distances = {8, 0, 3, 8, 12, 6, 8, 0, 48};
    //vector<int> correct_edit_distances = {8};
    //vector<int> correct_edit_distances = {0};
    //vector<int> correct_edit_distances = {48};
    
    vector<Alignment_t> alignments = genasm_gpu::align_all(reference, reads);

    if(alignments.size() != correct_edit_distances.size()){
        cout << "FAILED gpu_algorithm_correctness_test: align_all() produced wrong number of alignments" << endl;
        return;
    }

    bool success = true;
    for(int i = 0; i < alignments.size(); i++){
        if(alignments[i].edit_distance != correct_edit_distances[i]){
            cout << "FAILED gpu_algorithm_correctness_test: align_all() produced distance " << alignments[i].edit_distance;
            cout << " instead of " << correct_edit_distances[i];
            cout << " for read \""<< reads[i].description << "\"" << endl;
            success = false;
        }
        if(!validateCigarString(alignments[i], ref_begin, reads[i], reference)){
            success = false;
        }

    }
    if(success){
        cout << "PASSED gpu_algorithm_correctness_test" << endl;
    }
}

void cpu_algorithm_correctness_test(){
    Genome_t reference;
    reference.content = "AAAACCCCGGGGTTTT";
    
    CandidateLocation_t ref_begin;
    ref_begin.start_in_reference = 0;
    ref_begin.start_in_chromosome = 0;
    ref_begin.strand = true;
    ref_begin.chromosome = "";
    vector<CandidateLocation_t> ref_begin_vec(1, ref_begin);

    vector<Read_t> reads;
    reads.push_back({"test_read_4d12m4i",           "CCCCGGGGTTTTAAAA",         ref_begin_vec});
    reads.push_back({"test_read_16m",               "AAAACCCCGGGGTTTT",         ref_begin_vec});
    reads.push_back({"test_read_3d7m",              "ACCCCGG",                  ref_begin_vec});
    reads.push_back({"test_read_4m4d4m4i4m",        "AAAAGGGGAAAATTTT",         ref_begin_vec});
    reads.push_back({"test_read_12s4m",             "AAAAAAAAAAAAAAAA",         ref_begin_vec});
    reads.push_back({"test_read_1m1s1i3m1s2m3i",    "ATTAACGCCTTT",             ref_begin_vec});
    reads.push_back({"test_read_oversized",         "TTTTAAAACCCCGGGGTTTTAAAA", ref_begin_vec});
    reads.push_back({"test_read_empty",             "",                         ref_begin_vec});
    reads.push_back({"test_read_len64",             "TTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTTAAAACCCCGGGGTTTTAAAA", ref_begin_vec});

    vector<int> correct_edit_distances = {8, 0, 3, 8, 12, 6, 8, 0, 48};

    vector<Alignment_t> alignments = genasm_cpu::align_all(reference, reads);

    if(alignments.size() != correct_edit_distances.size()){
        cout << "FAILED cpu_algorithm_correctness_test: align_all() produced wrong number of alignments" << endl;
        return;
    }

    bool success = true;
    for(int i = 0; i < alignments.size(); i++){
        if(alignments[i].edit_distance != correct_edit_distances[i]){
            cout << "FAILED cpu_algorithm_correctness_test: align_all() produced distance " << alignments[i].edit_distance;
            cout << " instead of " << correct_edit_distances[i];
            cout << " for read \""<< reads[i].description << "\"" << endl;
            success = false;
        }
        if(!validateCigarString(alignments[i], ref_begin, reads[i], reference)){
            success = false;
        }

    }
    if(success){
        cout << "PASSED cpu_algorithm_correctness_test" << endl;
    }
}

void gpu_algorithm_performance_test(string reference_file_path, string reads_file_path, string alignments_file_path, int read_length_cap=-1, int dataset_inflation=1){
    if(enable_log) cerr << "Starting performance test..." << endl;

    if(enable_log) cerr << "Reading reference sequence..." << endl;
    Genome_t reference_genome = read_genome(reference_file_path);
    
    if(enable_log) cerr << "Reading reads files (~30 seconds)..." << endl;
    vector<Read_t> reads;
    read_fastq_and_seed_locations(reference_genome, reads_file_path, alignments_file_path, reads);

    if(enable_log) cerr << "Filtering reads..." << endl;
    //filter out any reverse complement reads
    for(Read_t &read : reads){
        read.locations.erase(remove_if(
                read.locations.begin(),
                read.locations.end(),
                [](CandidateLocation_t const &l){ return l.strand==false; }
            ),
            read.locations.end()
        );
    }

    //reads.erase(reads.begin()+1, reads.end());
    //reads.erase(reads.begin()+1000, reads.end());
    //reads = vector<Read_t>(115241, reads[0]);

    if(read_length_cap >= 0){
        for(Read_t &r: reads){
            r.content = r.content.substr(0, read_length_cap);
        }
    }

    if(dataset_inflation > 1){
        int old_size = reads.size();
        reads.resize(dataset_inflation*old_size);
        for(int i = 1; i < dataset_inflation; i++){
            copy_n(reads.begin(), old_size, reads.begin()+i*old_size);
        }
    }

    if(enable_log) cerr << "Sorting reads..." << endl;
    //sort reads in descending length
    sort(reads.begin(), reads.end(), [](Read_t &a, Read_t &b){return a.content.size() > b.content.size();});

    if(enable_log) cerr << "Running alignment algorithm..." << endl;

    vector<Alignment_t> alignments;
    long long core_algorithm_ns;
    auto workload = [&](){
        alignments = genasm_gpu::align_all(reference_genome, reads, &core_algorithm_ns);
    };
    long long end_to_end_ns = measure_ns(workload); //runtime including data transfer, conversion, readout, post-processing
    long long end_to_end_alignments_per_second = alignments.size() * 1000000000 / end_to_end_ns;
    long long core_algorithm_alignments_per_second = alignments.size() * 1000000000 / core_algorithm_ns;

    if(enable_log) cerr << "Sanity checking alignments..." << endl;
    
    size_t pair_idx = 0;
    for(Read_t &read : reads){
        for(CandidateLocation_t &location : read.locations){
            if(!validateCigarString(alignments[pair_idx], location, read, reference_genome)){
                cout << "FAILED sanity check in algorithm_performance_test ";
                cout << "for alignment " << pair_idx << endl;
            }
            pair_idx++;
        }
    }

    if(enable_log) cerr << "Done" << endl;

    cout << "align_all() took " << (end_to_end_ns/1000000) << "ms (data transfers, conversion, gpu kernel and post-processing)" << endl;
    cout << "GPU kernel took " << (core_algorithm_ns/1000000) << "ms" << endl;
    cout << "GPU kernel ran at " << core_algorithm_alignments_per_second << " aligns/second" << endl;
    //cout << "ran at " << end_to_end_alignments_per_second << " aligns/second" << endl;
}

void cpu_algorithm_performance_test(string reference_file_path, string reads_file_path, string alignments_file_path, int threads, int read_length_cap=-1, int dataset_inflation=1){
    if(enable_log) cerr << "Starting performance test..." << endl;

    if(enable_log) cerr << "Reading reference sequence..." << endl;
    Genome_t reference_genome = read_genome(reference_file_path);
    if(enable_log) cerr << "Reading reads files (~30 seconds)..." << endl;
    vector<Read_t> reads;
    read_fastq_and_seed_locations(reference_genome, reads_file_path, alignments_file_path, reads);

    if(enable_log) cerr << "Filtering reads..." << endl;
    //filter out any reverse complement reads
    for(Read_t &read : reads){
        read.locations.erase(remove_if(
                read.locations.begin(),
                read.locations.end(),
                [](CandidateLocation_t const &l){ return l.strand==false; }
            ),
            read.locations.end()
        );
    }
    //reads.erase(reads.begin()+1, reads.end());
    //reads.erase(reads.begin()+1000, reads.end());
    //reads = vector<Read_t>(115241, reads[0]);
    /*for(int i = 0; i < 115241 - 3; i+=4){
        reads[i+1] = reads[i];
        reads[i+2] = reads[i];
        reads[i+3] = reads[i];
    }*/
    if(read_length_cap >= 0){
        for(Read_t &r: reads){
            r.content = r.content.substr(0, read_length_cap);
        }
    }

    if(dataset_inflation > 1){
        int old_size = reads.size();
        reads.resize(dataset_inflation*old_size);
        for(int i = 1; i < dataset_inflation; i++){
            copy_n(reads.begin(), old_size, reads.begin()+i*old_size);
        }
    }

    if(enable_log) cerr << "Sorting reads..." << endl;
    //sort reads in descending length
    sort(reads.begin(), reads.end(), [](Read_t &a, Read_t &b){return a.content.size() > b.content.size();});

    if(enable_log) cerr << "Running alignment algorithm..." << endl;

    vector<Alignment_t> alignments;
    long long core_algorithm_ns;
    auto workload = [&](){
        alignments = genasm_cpu::align_all(reference_genome, reads, threads, &core_algorithm_ns);
    };
    long long end_to_end_ns = measure_ns(workload); //runtime including data transfer, conversion, readout, post-processing
    long long end_to_end_alignments_per_second = alignments.size() * 1000000000 / end_to_end_ns;
    long long core_algorithm_alignments_per_second = alignments.size() * 1000000000 / core_algorithm_ns;

    if(enable_log) cerr << "Sanity checking alignments..." << endl;
    
    size_t pair_idx = 0;
    for(Read_t &read : reads){
        for(CandidateLocation_t &location : read.locations){
            if(!validateCigarString(alignments[pair_idx], location, read, reference_genome)){
                cout << "FAILED sanity check in cpu_algorithm_performance_test ";
                cout << "for alignment " << pair_idx << endl;
            }
            pair_idx++;
        }
    }

    if(enable_log) cerr << "Done" << endl;

    cout << "align_all() took " << (end_to_end_ns/1000000) << "ms (data transfers, conversion, cpu kernel and post-processing)" << endl;
    cout << "CPU kernel took " << (core_algorithm_ns/1000000) << "ms" << endl;
    cout << "CPU kernel ran at " << core_algorithm_alignments_per_second << " aligns/second" << endl;
    //cout << "ran at " << end_to_end_alignments_per_second << " aligns/second" << endl;
}

void read_file_performance_test(string path){
    string raw_file;
    auto workload = [&path, &raw_file](){
        raw_file = read_file(path);
    };
    long long ns = measure_ns(workload);
    long long bytes_per_second = raw_file.size() * 1000000000 / ns;
    cout << "read_file() ran at " << bytes_per_second/1000000 << "MB/s in " << ns/1000000 << "ms"<< endl;
}

void read_genome_performance_test(string path){
    Genome_t reference_genome;
    auto workload = [&path, &reference_genome](){
        reference_genome = read_genome(path);
    };
    long long ns = measure_ns(workload);
    long long bytes_per_second = reference_genome.content.size() * 1000000000 / ns;
    cout << "read_genome() ran at " << bytes_per_second/1000000 << "MB/s in " << ns/1000000 << "ms"<< endl;
}

void read_fastq_performance_test(string path){
    vector<Read_t> reads;
    auto workload = [&path, &reads](){
        reads = read_fastq(path);
    };
    long long ns = measure_ns(workload);
    
    long long reads_bytes = 0;
    for(auto it = reads.begin(); it != reads.end(); it++){
        reads_bytes += it->content.size();
    }
    long long total_bytes = std::experimental::filesystem::file_size(path);

    long long bytes_per_second = total_bytes * 1000000000 / ns;
    cout << "read_fastq() ran at " << bytes_per_second/1000000 << "MB/s in " << ns/1000000 << "ms"<< endl;
}

void read_maf_performance_test(string path){
    vector<CandidateLocation_t> locations;
    auto workload = [&path, &locations](){
        locations = read_maf(path);
    };
    long long ns = measure_ns(workload);

    long long total_bytes = std::experimental::filesystem::file_size(path);
    long long bytes_per_second = total_bytes * 1000000000 / ns;
    cout << "read_maf() ran at " << bytes_per_second/1000000 << "MB/s in " << ns/1000000 << "ms"<< endl;
}

void read_fastq_and_seed_locations_performance_test(string fastq_path, string seeds_path){
    Genome_t dummy_genome;
    vector<Read_t> reads;
    auto workload = [&fastq_path, &seeds_path, &reads, &dummy_genome](){
        read_fastq_and_seed_locations(dummy_genome, fastq_path, seeds_path, reads);
    };
    long long ns = measure_ns(workload);
    
    long long reads_bytes = 0;
    for(auto it = reads.begin(); it != reads.end(); it++){
        reads_bytes += it->content.size();
    }
    long long total_bytes = reads_bytes*4; //approximate total file size

    long long bytes_per_second = total_bytes * 1000000000 / ns;
    cout << "read_fastq_and_seed_locations() ran at " << bytes_per_second/1000000 << "MB/s in " << ns/1000000 << "ms"<< endl;
}

void io_performance_test(string reference_file_path, string reads_file_path, string alignments_file_path){
    if(enable_log) cerr << "Starting IO performance test..." << endl;

    read_file_performance_test(reads_file_path);
    read_genome_performance_test(reference_file_path);
    read_fastq_performance_test(reads_file_path);
    read_maf_performance_test(alignments_file_path);
    read_fastq_and_seed_locations_performance_test(reads_file_path, alignments_file_path);
}

#define TWOBIT_AT(I, CHARARRAY) (((CHARARRAY)[(I)>>2] >> (6 - ((I%4)<<1))) & 0x03)
void print_twobit_as_ascii(long long length, vector<char> twobit){
    char *res = (char *)malloc(length + 1);
    for(int i = 0; i < length; i++){
        char code = TWOBIT_AT(i, twobit);

        if(code == 0x00) res[i] = 'A';
        if(code == 0x01) res[i] = 'C';
        if(code == 0x02) res[i] = 'G';
        if(code == 0x03) res[i] = 'T';
    }
    res[length] = '\0';
    cout << res << endl;
    free(res);
}

void ascii_to_two_bit_correctness_test(){
    vector<string> inputs = {
        "",
        "A",
        "ACGT",
        "ACGTA",
        "AAAAAAAACCCCCCCCGGGGGGGGTTTTTTTT",
        "AAAAAAAACCCCCCCCGGGGGGGGTTTTTTTTA"
    };

    vector<vector<char>> correct_results;
    for(int i = 0; i < inputs.size(); i++){
        vector<char> res;
        for(int quad = 0; quad*4 < inputs[i].size(); quad++){
            char twobit = 0x00;
            for(int sub_idx = 0; sub_idx < 4 && quad*4 + sub_idx < inputs[i].size(); sub_idx++){
                char c = inputs[i][quad*4 + sub_idx];
                char code;
                if(c == 'A') code = 0x00;
                if(c == 'C') code = 0x01;
                if(c == 'G') code = 0x02;
                if(c == 'T') code = 0x03;
                twobit |= code << (6 - 2*sub_idx);
            }
            res.push_back(twobit);
        }
        correct_results.push_back(res);
    }

    char **ascii_strings, **twobit_strings;
    long long *string_lengths;
    hipMallocManaged(&ascii_strings, sizeof(char *) * inputs.size());
    hipMallocManaged(&twobit_strings, sizeof(char *) * inputs.size());
    hipMallocManaged(&string_lengths, sizeof(long long) * inputs.size());

    for(int i = 0; i < inputs.size(); i++){
        hipMallocManaged(ascii_strings + i, sizeof(char)*inputs[i].size());
        hipMallocManaged(twobit_strings + i, sizeof(char)*(inputs[i].size()+3)/4);
        string_lengths[i] = inputs[i].size();
        for(int j = 0; j < inputs[i].size(); j++){
            ascii_strings[i][j] = inputs[i][j];
        }
    }

    genasm_gpu::ascii_to_twobit_strings<<<32, 32>>>(inputs.size(), string_lengths, ascii_strings, twobit_strings);
    hipDeviceSynchronize();

    bool success = true;
    for(int i = 0; i < inputs.size(); i++){
        bool equal = true;
        for(int j = 0; j < correct_results[i].size(); j++){
            if(twobit_strings[i][j] != correct_results[i][j]) equal = false;
        }
        if(!equal){
            cout << "FAILED ascii_to_two_bit_correctness_test: produced twobit string" << endl;
            print_twobit_as_ascii(inputs[i].size(), vector<char>(twobit_strings[i], twobit_strings[i] + correct_results[i].size()));
            cout << "instead of" << endl;
            print_twobit_as_ascii(inputs[i].size(), correct_results[i]);
            cout << " for index " << i << endl << endl;
            success = false;
        }
    }
    if(success){
        cout << "PASSED ascii_to_two_bit_correctness_test" << endl;
    }
}

void ascii_to_two_bit_performance_test(string reads_file_path){    
    vector<Read_t> reads = read_fastq(reads_file_path);
    //reads.erase(reads.begin()+1000, reads.end());


    vector<vector<char>> correct_results;
    for(int i = 0; i < reads.size(); i++){
        vector<char> res;
        for(int quad = 0; quad*4 < reads[i].content.size(); quad++){
            char twobit = 0x00;
            for(int sub_idx = 0; sub_idx < 4 && quad*4 + sub_idx < reads[i].content.size(); sub_idx++){
                char c = reads[i].content[quad*4 + sub_idx];
                char code;
                if(c == 'A') code = 0x00;
                if(c == 'C') code = 0x01;
                if(c == 'G') code = 0x02;
                if(c == 'T') code = 0x03;
                twobit |= code << (6 - 2*sub_idx);
            }
            res.push_back(twobit);
        }
        correct_results.push_back(res);
    }

    long long int total_ascii_length = 0;
    long long int total_twobit_length = 0;
    for(int i = 0; i < reads.size(); i++){
        total_ascii_length += reads[i].content.size();
        total_twobit_length += (reads[i].content.size()+3)/4;
    }

    char **ascii_strings, **twobit_strings;
    long long *string_lengths;
    char *ascii_block, *twobit_block;
    hipMallocManaged(&ascii_strings, sizeof(char *) * reads.size());
    hipMallocManaged(&twobit_strings, sizeof(char *) * reads.size());
    hipMallocManaged(&string_lengths, sizeof(long long) * reads.size());
    hipMallocManaged(&ascii_block, sizeof(char) * total_ascii_length);
    hipMallocManaged(&twobit_block, sizeof(char) * total_twobit_length);
    char *next_ascii_start = ascii_block;
    char *next_twobit_start = twobit_block;

    for(int i = 0; i < reads.size(); i++){
        ascii_strings[i] = next_ascii_start;
        twobit_strings[i] = next_twobit_start;
        next_ascii_start += reads[i].content.size();
        next_twobit_start += (reads[i].content.size()+3)/4;
        string_lengths[i] = reads[i].content.size();
        for(int j = 0; j < reads[i].content.size(); j++){
            ascii_strings[i][j] = reads[i].content[j];
        }
    }

    auto workload = [&](){
        genasm_gpu::ascii_to_twobit_strings<<<256, 32>>>(reads.size(), string_lengths, ascii_strings, twobit_strings);
        hipDeviceSynchronize();
    };
    long long ns = measure_ns(workload);
    long long reads_per_second = reads.size() * 1000000000 / ns;
    cout << "ascii_to_two_bit_performance_test" << endl;
    cout << "ran at " << reads_per_second << " reads/second" << endl;

    for(int i = 0; i < reads.size(); i++){
        bool equal = true;
        for(int j = 0; j < correct_results[i].size(); j++){
            if(twobit_strings[i][j] != correct_results[i][j]) equal = false;
        }
        if(!equal){
            cout << "ERROR in ascii_to_two_bit_performance_test: produced twobit string" << endl;
            print_twobit_as_ascii(reads[i].content.size(), vector<char>(twobit_strings[i], twobit_strings[i] + correct_results[i].size()));
            cout << "instead of" << endl;
            print_twobit_as_ascii(reads[i].content.size(), correct_results[i]);
            cout << " for index " << i << endl << endl;
        }
    }
}

void parse_args(int argc, char **argv, string &reference_file, string &reads_file, string &seeds_file, bool &gpu_info_only, bool &verbose, bool &unit_tests, bool &cpu_performance_test){
    //default values
    reference_file = "datasets/human_genome/pacbio-chr1-simulated-m10k-k5_0001.ref";
    reads_file =     "datasets/human_genome/pacbio-chr1-simulated-m10k-k5_0001.fastq";
    seeds_file =     "datasets/human_genome/pacbio-chr1-simulated-m10k-k5_0001.maf";


    gpu_info_only =         OPT_EXISTS == get_cmd_option(argc, argv, "--gpu_info_only");
    verbose =               OPT_EXISTS == get_cmd_option(argc, argv, "--verbose");
    unit_tests =            OPT_EXISTS == get_cmd_option(argc, argv, "--unit_tests");
    cpu_performance_test =  OPT_EXISTS == get_cmd_option(argc, argv, "--cpu_performance_test");

    bool help_and_exit = false;
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--reference", reference_file);
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--reads", reads_file);
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--seeds", seeds_file);
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--gpu_info_only");
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--verbose");
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--unit_tests");
    help_and_exit |= OPT_INVALID == get_cmd_option(argc, argv, "--cpu_performance_test");
    help_and_exit |= OPT_MISSING != get_cmd_option(argc, argv, "--help");
    help_and_exit |= !check_options(argc, argv, {"--reference", "--reads", "--seeds", "--help", "--gpu_info_only", "--verbose", "--unit_tests", "--cpu_performance_test"});

    string help_text =
        "tests[.exe] [options]\n"
        "Options:\n"
        "--reference=[path to reference FASTA] -- overide default reference data for performance test\n"
        "--reads=[path to reads FASTQ]         -- overide default reads data for performance test\n"
        "--seeds=[path to MAF or PAF]          -- overide default seeds data for performance test\n"
        "--gpu_info_only                       -- only print GPU info\n"
        "--verbose                             -- print progress to stderr. Otherwise, only test results are printed\n"
        "--unit_tests                          -- run unit tests (default: disabled)\n"
        "--cpu_performance_test                -- run cpu algorithm performance test (default: gpu)\n"
        "--help                                -- displays this information\n";

    if(help_and_exit){
        cout << help_text << flush;
        exit(0);
    }
}

void print_gpu_info(){
    int nDevices;
    hipGetDeviceCount(&nDevices);
    cout << nDevices << " visible GPU(s):" << endl;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "idx=" << i;
        cout << " name=\"" << prop.name  << "\"";
        cout << " SMs=" << prop.multiProcessorCount;
        cout << " smem=" << (prop.sharedMemPerMultiprocessor >> 10) << "kiB" << endl;
    }
    cout << endl;
}

int main(int argc, char **argv){
    string reference_file, reads_file, seeds_file;
    bool gpu_info_only, verbose, unit_tests, cpu_performance_test;
    parse_args(argc, argv, reference_file, reads_file, seeds_file, gpu_info_only, verbose, unit_tests, cpu_performance_test);
    if(gpu_info_only){
        print_gpu_info();
        exit(0);
    }

    genasm_cpu::enabled_algorithm_log = verbose;
    genasm_gpu::enabled_algorithm_log = verbose;
    enable_log = verbose;

    if(unit_tests){
        print_gpu_info();
        bitvector_tests();
        //io_performance_test(datasets_dir);
        ascii_to_two_bit_correctness_test();
        //ascii_to_two_bit_performance_test(reads_file);
        cpu_algorithm_correctness_test();
        gpu_algorithm_correctness_test();
    }
    else{
        if(cpu_performance_test){
            cpu_algorithm_performance_test(reference_file, reads_file, seeds_file, 32);
        }
        else{
            print_gpu_info();
            gpu_algorithm_performance_test(reference_file, reads_file, seeds_file);
        }
    }

    return 0;
}
